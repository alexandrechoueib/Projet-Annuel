#include "devices.h"
 
using namespace ez::cume;

Devices *Devices::_instance = NULL;

Devices::Devices() {
	cume_check( hipGetDeviceCount(&devices_count) );
	devices = new hipDeviceProp_t[ devices_count ];
    for (integer i = 0; i < devices_count; ++i) {
		cume_check( hipGetDeviceProperties(&devices[i], i) );
    }
}

Devices& Devices::instance() {
	if (_instance == NULL) {
		_instance = new Devices;
	}
	return *_instance;
}
	
Devices::~Devices() {
	delete [] devices;
}
	
void Devices::select(integer device_id) {
	ensure((device_id >= 0) && (device_id < devices_count));
	cume_check( hipSetDevice(device_id) );
}

hipDeviceProp_t *Devices::device_properties(integer device_id) {
	ensure((device_id >= 0) && (device_id < devices_count));
	return &devices[device_id];
}

void Devices::memory_report(ostream& out) {
	size_t free_byte;
    size_t total_byte;
    cume_check( hipMemGetInfo( &free_byte, &total_byte) );
    double free_db = static_cast<double>(free_byte);
    double total_db = static_cast<double>(total_byte);
    double used_db = total_db - free_db ;
	const double one_mb = 1024.0*1024.0;
    out << "GPU memory usage: used = " << (used_db/one_mb)
            << ", free = " << (free_db/one_mb)
            << ", total = " << (total_db/one_mb) << endl;
}


ostream& Devices::print(ostream& out) {
	for (integer i=0; i<devices_count; ++i) {
		out << "device " << i << ": " << devices[i].name;
		out << " " << devices[i].totalGlobalMem << " bytes";
		natural mem = devices[i].totalGlobalMem / (1024*1024);
		out << " (" << mem << " Mb)";   
		out << endl;
	}
	return out;
}
