#include "timer.h"

using namespace ez::cume;

Timer::Timer() {
	hipEventCreate(&t_start);
	hipEventCreate(&t_stop);
}

/**
 * destructor
 */
Timer::~Timer() {
	hipEventDestroy(t_start);
	hipEventDestroy(t_stop);
}

/**
 * start timer
 */
void Timer::start() {
	hipEventRecord(t_start, 0);
}

/**
 * stop timer
 */
void Timer::stop() {
	hipEventRecord(t_stop, 0);
	hipEventSynchronize(t_stop);
}	

float Timer::elapsed() {
	float milli_seconds = 0.0f;
	hipEventElapsedTime(&milli_seconds, t_start, t_stop);
	return milli_seconds;
}

/**
 * print timer difference in milliseconds
 */
ostream& Timer::print(ostream& out) {
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, t_start, t_stop);
	out.setf(ios::fixed);
	out.precision(2);
	out << elapsed_time << "ms";
	return out;
}
